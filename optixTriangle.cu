#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "optixTriangle.h"
#include <cuda/helpers.h>

#include <sutil/vec_math.h>
#define lights_num 2

extern "C" {
__constant__ Params params;
}
__constant__ float3 lights[lights_num] = {{-1.0f, 0.0f, 1.0f},{1.0f, 0.0f, 1.0f}};



static __forceinline__ __device__ void setPayload( float3 p )
{
    optixSetPayload_0( __float_as_uint( p.x ) );
    optixSetPayload_1( __float_as_uint( p.y ) );
    optixSetPayload_2( __float_as_uint( p.z ) );
}


static __forceinline__ __device__ void computeRay( uint3 idx, uint3 dim, float3& origin, float3& direction )
{
    const float3 U = params.cam_u;
    const float3 V = params.cam_v;
    const float3 W = params.cam_w;
    const float2 d = 2.0f * make_float2(
            static_cast<float>( idx.x ) / static_cast<float>( dim.x ),
            static_cast<float>( idx.y ) / static_cast<float>( dim.y )
            ) - 1.0f;

    origin    = params.cam_eye;
    direction = normalize( d.x * U + d.y * V + W );
}


extern "C" __global__ void __raygen__rg()
{
    // Lookup our location within the launch grid
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    // Map our launch idx to a screen location and create a ray from the camera
    // location through the screen
    float3 ray_origin, ray_direction;
    computeRay( idx, dim, ray_origin, ray_direction );

    // Trace the ray against our scene hierarchy
    unsigned int p0, p1, p2;
    optixTrace(
            params.handle,
            ray_origin,
            ray_direction,
            0.0f,                // Min intersection distance
            1e16f,               // Max intersection distance
            0.0f,                // rayTime -- used for motion blur
            OptixVisibilityMask( 255 ), // Specify always visible
            OPTIX_RAY_FLAG_NONE,
            0,                   // SBT offset   -- See SBT discussion
            1,                   // SBT stride   -- See SBT discussion
            0,                   // missSBTIndex -- See SBT discussion
            p0, p1, p2 );
    float3 result;
    result.x = __uint_as_float( p0 );
    result.y = __uint_as_float( p1 );
    result.z = __uint_as_float( p2 );

    // Record results in our output raster
    params.image[idx.y * params.image_width + idx.x] = make_color( result );
}


extern "C" __global__ void __miss__ms()
{
    MissData* miss_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    
    
    setPayload(miss_data->bg_color);
}


extern "C" __global__ void __closesthit__ch()
{
    // When built-in triangle intersection is used, a number of fundamental
    // attributes are provided by the OptiX API, indlucing barycentric coordinates.
    const float2 barycentrics = optixGetTriangleBarycentrics();
    unsigned int index = optixGetPrimitiveIndex();
    float3 data[3];
    optixGetTriangleVertexData(params.handle, index, 0, 0, data);

    
    float3 world_position = optixGetWorldRayOrigin() + optixGetWorldRayDirection() * optixGetRayTmax();
    float result = 100.f;
    for (int i = 0; i < lights_num; i++)
    {
        float distance = sqrt((world_position.x - lights[i].x) * (world_position.x - lights[i].x) +
            (world_position.y - lights[i].y) * (world_position.y - lights[i].y) +
            (world_position.z - lights[i].z) * (world_position.z - lights[i].z));
        float3 direction = { lights[i].x - world_position.x,lights[i].y - world_position.y, lights[i].z - world_position.z };
        float db = 100.f;
        float tmin = 0.f;
        int n = 0;
        while ((tmin < distance)) {
            unsigned int p0, p1, p2;
            n++;
            optixTrace(
                params.handle,
                world_position,
                normalize(direction),
                tmin,                // Min intersection distance
                distance,            // Max intersection distance
                0.0f,                // rayTime -- used for motion blur
                OptixVisibilityMask(254), // Specify always visible
                OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                1,                   // SBT offset   -- See SBT discussion
                1,                   // SBT stride   -- See SBT discussion
                1,                   // missSBTIndex -- See SBT discussion
                p0, p1, p2);
            db -= __uint_as_float(p0);
            tmin = __uint_as_float(p1);
        }
        if (db < result) result = db;
        result = n * 10;
    }
    float red = (result)/100.f;
    if (red < 0.) red = 0;
    setPayload(make_float3(red, 0, 0));


    /*if (barycentrics.y < 0.03  || barycentrics.x < 0.03 || barycentrics.x + barycentrics.y > 0.97)
    {
        setPayload(make_float3(0, 0, 0));
    }
    else
    {
        setPayload(make_float3(1. / (distance * distance), 0, 0));
    }*/
    //setPayload(make_float3(1, 0, 0));
}
extern "C" __global__ void __closesthit__ch_sh()
{
    float3 origin = optixGetWorldRayOrigin();
    float tmax = optixGetRayTmax();
    float3 world_position = origin + optixGetWorldRayDirection() * tmax;
    float distance = sqrt((world_position.x - origin.x) * (world_position.x - origin.x) +
        (world_position.y - origin.y) * (world_position.y - origin.y) +
        (world_position.z - origin.z) * (world_position.z - origin.z)) - optixGetRayTmin();
    setPayload(make_float3(distance * 100.f, tmax, 0));
}

extern "C" __global__ void __miss__ms_sh()
{
    float3 origin = optixGetWorldRayOrigin();
    float tmax = optixGetRayTmax();
    float3 world_position = origin + optixGetWorldRayDirection() * tmax;
    float distance = sqrt((world_position.x - origin.x) * (world_position.x - origin.x) +
        (world_position.y - origin.y) * (world_position.y - origin.y) +
        (world_position.z - origin.z) * (world_position.z - origin.z)) - optixGetRayTmin();
    setPayload(make_float3(distance * 100, tmax, 0));
}
